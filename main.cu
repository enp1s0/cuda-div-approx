#include "hip/hip_runtime.h"
// Ref : https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#floating-point-instructions-div
#include <iostream>
#include <cmath>
#include <cutf/memory.hpp>

constexpr unsigned block_size = 1u << 8;

// Div operations
struct approx_div;
struct full_range_approx_div;
struct ieee_div;

template <class DivOp>
__device__ float div(const float a, const float b);

template <>
__device__ float div<approx_div>(const float a, const float b) {
	float r;
	asm(
			R"(
{
div.approx.f32 %0, %1, %2;
}
)": "=f"(r) : "f"(a), "f"(b)
			);
	return r;
}

template <>
__device__ float div<full_range_approx_div>(const float a, const float b) {
	float r;
	asm(
			R"(
{
div.full.f32 %0, %1, %2;
}
)": "=f"(r) : "f"(a), "f"(b)
			);
	return r;
}

template <>
__device__ float div<ieee_div>(const float a, const float b) {
	float r;
	asm(
			R"(
{
div.rn.f32 %0, %1, %2;
}
)": "=f"(r) : "f"(a), "f"(b)
			);
	return r;
}

template <class DivOp>
__global__ void div_kernel(
		double* const r_ptr,
		const float* const b_ptr,
		const unsigned array_length
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= array_length) return;

	const auto a = 1. / 3;
	const auto b = b_ptr[tid];

	const auto dp_result = a / b;
	const auto sp_result = div<DivOp>(a, b);

	r_ptr[tid] = abs(dp_result - sp_result) / dp_result;
}

void test(
		const int exp_min,
		const int exp_max,
		const unsigned num_mantissa_split
		) {
	const auto array_length = (exp_max - exp_min + 1) * num_mantissa_split;
	auto b_array = cutf::memory::get_host_unique_ptr<float>(array_length);

	for (int e = exp_min; e <= exp_max; e++) {
		for (unsigned i = 0; i < num_mantissa_split; i++) {
			const auto mantissa_dp = static_cast<double>(1) * i / num_mantissa_split + 1;
			b_array.get()[(e - exp_min) * num_mantissa_split + i] = mantissa_dp * std::pow(2., static_cast<double>(e));
		}
	}

	auto approx_result = cutf::memory::get_host_unique_ptr<double>(array_length);
	div_kernel<approx_div><<<(array_length + block_size - 1) / block_size, block_size>>>(
			approx_result.get(),
			b_array.get(),
			array_length
			);
	auto full_range_approx_result = cutf::memory::get_host_unique_ptr<double>(array_length);
	div_kernel<full_range_approx_div><<<(array_length + block_size - 1) / block_size, block_size>>>(
			full_range_approx_result.get(),
			b_array.get(),
			array_length
			);
	auto ieee_result = cutf::memory::get_host_unique_ptr<double>(array_length);
	div_kernel<ieee_div><<<(array_length + block_size - 1) / block_size, block_size>>>(
			ieee_result.get(),
			b_array.get(),
			array_length
			);
	hipDeviceSynchronize();

	std::printf("b,approx,full_range_approx,ieee\n");
	for (int e = exp_min; e <= exp_max; e++) {
		for (unsigned i = 0; i < num_mantissa_split; i++) {
			const auto array_index = (e - exp_min) * num_mantissa_split + i;
			std::printf("%e,%e,%e,%e\n",
					b_array.get()[array_index],
					approx_result.get()[array_index],
					full_range_approx_result.get()[array_index],
					ieee_result.get()[array_index]
					);
		}
	}
}

int main() {
	test(-126, 126, 10);
}
